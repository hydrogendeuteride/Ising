#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <fstream>

#define L 512
#define THREADS_PER_BLOCK 16
#define BLOCKS_PER_GRID (L / THREADS_PER_BLOCK)
#define MC_STEPS 1000

__global__ void setupKernel(hiprandState *state, unsigned long seed)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = x + y * L;

    if (x < L && y < L)
    {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

__global__ void metropolis(int *spin, hiprandState *state, float T, int parity)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= L || y >= L)
        return;

    if ((x + y) % 2 != parity)
        return;

    int idx = x + y * L;

    int left = ((x - 1 + L) % L) + y * L;
    int right = ((x + 1) % L) + y * L;
    int up = x + ((y - 1 + L) % L) * L;
    int down = x + ((y + 1) % L) * L;

    int sumNeighbors = spin[left] + spin[right] + spin[up] + spin[down];
    int deltaE = 2 * spin[idx] * sumNeighbors;

    hiprandState localState = state[idx];
    float randVal = hiprand_uniform(&localState);
    state[idx] = localState;

    if (deltaE <= 0 || randVal < expf(-deltaE / T))
    {
        spin[idx] = -spin[idx];
    }
}

int main()
{
    int *spinHost = (int *) malloc(L * L * sizeof(int));
    int *spinDev;

    hiprandState *devStates;

    srand(time(NULL));
    for (int i = 0; i < L * L; ++i)
    {
        spinHost[i] = (rand() % 2) * 2 - 1;
    }

    hipMalloc((void **) &spinDev, L * L * sizeof(int));
    hipMalloc((void **) &devStates, L * L * sizeof(hiprandState));

    hipMemcpy(spinDev, spinHost, L * L * sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS_PER_GRID, BLOCKS_PER_GRID);
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    setupKernel<<<blocks, threads>>>(devStates, time(NULL));

    float T = 2.269;

    for (int step = 0; step < MC_STEPS; ++step)
    {
        metropolis<<<blocks, threads>>>(spinDev, devStates, T, 0);
        hipDeviceSynchronize();

        metropolis<<<blocks, threads>>>(spinDev, devStates, T, 1);
        hipDeviceSynchronize();
    }

    hipMemcpy(spinHost, spinDev, L * L * sizeof(int), hipMemcpyDeviceToHost);

    std::ofstream outFile("spins.txt");
    for (int i = 0; i < L; ++i)
    {
        for (int j = 0; j < L; ++j)
        {
            outFile << spinHost[i * L + j] << " ";
        }
        outFile << std::endl;
    }
    outFile.close();

    int sum = 0;
    for (int i = 0; i < L * L; i++)
    {
        sum += spinHost[i];
    }

    float m_avg = (float) sum / (L * L);
    std::cout << "Average magnetization: " << m_avg << std::endl;

    hipFree(spinDev);
    hipFree(devStates);
    free(spinHost);

    return 0;
}
